
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <random>

#define SIZE (1<<30)
#define THREADS 64

using namespace std;

__global__ void quicksort(int *data, int start, int end, int threads) {
    if(start >= end) {
        return;
    }

    int pivot = start;
    int i = start + 1;
    int j = end - 1;

    while(i <= j) {
        while(data[i] < data[pivot]) i++;
        while(data[j] > data[pivot]) j--;

        if(i <= j) {
            int temp = data[i];
            data[i] = data[j];
            data[j] = temp;
            i++;
            j--;
        }
    }

    int temp = data[pivot];
    data[pivot] = data[j];
    data[j] = temp;

    if(i < end) {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort<<<1, threads, 0, s>>>(data, i, end, (end - i));
        hipStreamDestroy(s);
    }

    if(j > start) {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort<<<1, threads, 0, s1>>>(data, start, j, (j - start));
        hipStreamDestroy(s1);
    }
}


int main() {
    random_device rd;
    mt19937 random(rd());
    uniform_int_distribution<> bounds{1, 10};

    int *h_arr, *d_arr;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
        
    h_arr = new int[SIZE];

    cout << "Filling array with values...";
        
    for(int *ptr = h_arr; ptr < h_arr + SIZE; ptr++) {
        *ptr = bounds(random);
    }
    
    cout << "Finished." << endl;

    cout << "Sorting..." << endl;     
     
    // for(int i = 0; i < SIZE; i++) {
    //   cout << h_arr[i] << " ";
    // }
    // cout << endl;
    
    hipMalloc(&d_arr, SIZE * sizeof(int));
    
    hipMemcpy(d_arr, h_arr, SIZE * sizeof(int), hipMemcpyHostToDevice);
        
    hipEventRecord(start);
    quicksort<<<1, 1>>>(d_arr, 0, SIZE, THREADS);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    
    hipMemcpy(h_arr, d_arr, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    if(ms / 1000 > 1) {
      cout << "Time execution: " << ms/1000 << "s" << endl;
    } else {
      cout << "Time execution: " << ms << "ms" << endl;
    }

    cout << "Array size: " << SIZE;
    
    

    // for(int i = 0; i < SIZE; i++) {
    //   cout << h_arr[i] << " ";
    // }

    delete[] h_arr;
    hipFree(d_arr);

    return 0;
}